#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include "simulator.h"
 #include "model.h"
 //#include "realm/runtime_impl.h"
 //#include "realm/cuda/cuda_module.h"
 #include "cuda_helper.h"
 
 typedef long long int coord_t;
 
 typedef Realm::Point<1, coord_t> Point1;
 typedef Realm::Rect<1, coord_t> Rect1;
 
 Simulator::Simulator(const FFModel* model,
                      FFHandler _handler,
                      Memory _memory,
                      MachineModel *machine)
 : memory(_memory), handler(_handler),
   offset(0), warmup_times(5), repeat_times(10),
   computationMode(model->config.computationMode)
 {
   // Allocate simulator memory
   Rect1 bounds(Point1(0), Point1(0));
   std::vector<size_t> field_sizes;
   field_sizes.push_back(model->config.simulator_work_space_size);
   Realm::RegionInstance::create_instance(simulatorInst,
       memory, bounds, field_sizes, 0, Realm::ProfilingRequestSet()).wait();
   base_ptr = (char*)simulatorInst.pointer_untyped(0, sizeof(char));
   capacity = model->config.simulator_work_space_size;
 
   if (model->simonly) {
     // allocate memory for workspace
     Memory gpu_mem = _memory;
     Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
         Realm::Point<1, coord_t>(handler.workSpaceSize-1));
     std::vector<size_t> field_sizes;
     field_sizes.push_back(sizeof(char));
     Realm::RegionInstance workspaceInst;
     Realm::RegionInstance::create_instance(workspaceInst, gpu_mem, bounds,
         field_sizes, 0, Realm::ProfilingRequestSet()).wait();
     handler.workSpace = workspaceInst.pointer_untyped(0, sizeof(char));
   }
 
   size_t max_num_tasks = 256 * 1024 * 1024;
 
   hipEventCreate(&start_event);
   hipEventCreate(&end_event);
   conv2d_meta = new Conv2DMeta(handler);
   linear_meta = new LinearMeta(handler, 4096);
   pool2d_meta = new Pool2DMeta(handler);
   ele_unary_meta = new ElementUnaryMeta(handler);
   ele_binary_meta = new ElementBinaryMeta(handler);
   //softmax_meta = new SoftmaxMeta(handler);
   batch_matmul_meta = new BatchMatmulMeta(handler);
   concat_meta = new ConcatMeta(handler);
   //dropout_meta = new DropoutMeta(handler);
   transpose_meta = new TransposeMeta(handler);
   this->machine = machine;
   segment_size = model->config.simulator_segment_size;
   max_num_segments = model->config.simulator_max_num_segments;
   // Initialize task manager
   task_manager = new TaskManager(max_num_tasks);
 
   measurements = nullptr;
   l1optimizer = nullptr;
 }
 
 Simulator::~Simulator(void)
 {
   simulatorInst.destroy();
 }
 
 __host__
 void Simulator::strategy_search_task(const Task *task,
                                      const std::vector<PhysicalRegion> &regions,
                                      Context ctx, Runtime *runtime)
 {
   const FFModel* model = *((FFModel**) task->args);
   Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
          .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
   // Realm::MemoryImpl* memImpl =
   //     Realm::get_runtime()->get_memory_impl(gpu_mem);
   // Realm::Cuda::GPUFBMemory* memFBImpl = (Realm::Cuda::GPUFBMemory*) memImpl;
   // off_t offset = memFBImpl->alloc_bytes_local(model->config.simulator_work_space_size);
   // void* base_ptr = memFBImpl->get_direct_ptr(offset, 0);
   MachineModel *machine;
   if (model->config.machine_model_version == 0) {
     machine = (MachineModel *) new SimpleMachineModel(model->config.numNodes, model->config.workersPerNode, gpu_mem.capacity());
   }
   else if (model->config.machine_model_version == 1 and !model->config.machine_model_file.empty()) {
     machine = (MachineModel *) new EnhancedMachineModel(model->config.machine_model_file, gpu_mem.capacity());
   }
   else {
     assert(false && "machine model creation error: currently only support machine-model-version = 0 or 1. When machine-model-version = 1, machine-model-file should not be empty.");
   }
   // Assume this task is running on GPU0
   Simulator* simulator = new Simulator(model, model->handlers[0], gpu_mem, machine);
   // Set cublas/cudnn streams to allow Realm catch the events
 
   hipStream_t stream;
   checkCUDA(get_legion_stream(&stream));
   checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
   checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
 
   std::map<Op*, ParallelConfig> strategies;
   if (model->config.import_strategy_file.length() > 0) {
     // Load the strategy from config.strategies
     for (size_t l = 0; l < model->layers.size(); l++) {
       MappingTagID key = FFConfig::get_hash_id(std::string(model->layers[l]->name));
       std::map<MappingTagID, ParallelConfig>::const_iterator iter;
       iter = model->config.strategies.find(key);
       if (iter == model->config.strategies.end()) {
         fprintf(stderr, "ERROR: Cannot find strategy for operator %s in "
                 "strategy file %s\n", model->layers[l]->name,
                 model->config.import_strategy_file.c_str());
       }
       strategies[model->layers[l]] = iter->second;
     }
   } else {
     // Start from data parallel
     for (size_t l = 0; l < model->layers.size(); l++) {
       strategies[model->layers[l]] = model->layers[l]->get_data_parallel_config(*model);
     }
   }
   if (model->config.computationMode == COMP_MODE_TRAINING) {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(TRAINING)\n",
         model->config.search_budget, model->config.search_alpha);
   } else {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(INFERENCE)\n",
         model->config.search_budget, model->config.search_alpha);
   }
   model->optimize(simulator, strategies, model->config.search_budget,
       model->config.search_alpha, model->config.computationMode, model->config.enable_propagation);
   if (model->config.export_strategy_file.length() > 0) {
     fprintf(stderr, "Exporting the best discovered strategy to %s.\n",
         model->config.export_strategy_file.c_str());
     std::map<Op*, ParallelConfig>::const_iterator iter;
     std::map<std::string, ParallelConfig> strategy_output;
     for (iter = strategies.begin(); iter != strategies.end(); iter++) {
       strategy_output[iter->first->name] = iter->second;
     }
     save_strategies_to_file(model->config.export_strategy_file, strategy_output);
     fprintf(stderr, "To use the strategy for distributed training, restart"
         " FlexFlow and import the strategy (i.e., --import %s)\n",
         model->config.export_strategy_file.c_str());
     exit(0);
   }  else {
     fprintf(stderr, "The best discovered strategy is not exported.\n"
         "Please set a path to export the strategy using --export or --export-strategy.\n");
     exit(0);
   }
   // Start from data
   // memFBImpl->free_bytes_local(offset, model->config.simulator_work_space_size);
   delete(simulator);
   delete(machine);
 }
 
 __host__
 void Simulator::simulation_task(const Task *task,
                                      const std::vector<PhysicalRegion> &regions,
                                      Context ctx, Runtime *runtime)
 {
   FFModel* model = *((FFModel**) task->args);
   Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
          .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
   // Realm::MemoryImpl* memImpl =
   //     Realm::get_runtime()->get_memory_impl(gpu_mem);
   // Realm::Cuda::GPUFBMemory* memFBImpl = (Realm::Cuda::GPUFBMemory*) memImpl;
   // off_t offset = memFBImpl->alloc_bytes_local(model->config.simulator_work_space_size);
   // void* base_ptr = memFBImpl->get_direct_ptr(offset, 0);
   BigSwitchNetworkTopologyGenerator topo_gen = BigSwitchNetworkTopologyGenerator(model->config.numNodes);
   
   // NetworkedMachineModel *nmachine = new NetworkedMachineModel(model->config.numNodes, 
   //   model->config.workersPerNode,  
   //   1,
   //   model->config.network_latency,
   //   topo_gen.generate_topology(),
   //   gpu_mem.capacity(),
   //   20.0 * 1024 * 1024 / 8
   // );
   // nmachine->set_pcie(false);
   // nmachine->set_pipeline(false);
 
   SimpleMachineModel* nmachine = new SimpleMachineModel(model->config.numNodes, model->config.workersPerNode, gpu_mem.capacity());
 
   MachineModel *machine;
   machine = reinterpret_cast<MachineModel*>(nmachine);
 
   // Assume this task is running on GPU0
   Simulator* simulator = new Simulator(model, model->handlers[0], gpu_mem, machine);
   // Set cublas/cudnn streams to allow Realm catch the events
 
   if (model->config.mfile != "") {
     model->load_measurement(simulator, model->config.mfile);
   }
 
   hipStream_t stream;
   checkCUDA(get_legion_stream(&stream));
   checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
   checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
 
   std::map<Op*, ParallelConfig> strategies;
   if (model->config.import_strategy_file.length() > 0) {
     // Load the strategy from config.strategies
     for (size_t l = 0; l < model->layers.size(); l++) {
       MappingTagID key = FFConfig::get_hash_id(std::string(model->layers[l]->name));
       std::map<MappingTagID, ParallelConfig>::const_iterator iter;
       iter = model->config.strategies.find(key);
       if (iter == model->config.strategies.end()) {
         fprintf(stderr, "ERROR: Cannot find strategy for operator %s in "
                 "strategy file %s\n", model->layers[l]->name,
                 model->config.import_strategy_file.c_str());
       }
       strategies[model->layers[l]] = iter->second;
     }
   } else {
     // Start from data parallel
     for (size_t l = 0; l < model->layers.size(); l++) {
       strategies[model->layers[l]] = model->layers[l]->get_data_parallel_config(*model);
     }
   }
   if (model->config.computationMode == COMP_MODE_TRAINING) {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(TRAINING)\n",
         model->config.search_budget, model->config.search_alpha);
   } else {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(INFERENCE)\n",
         model->config.search_budget, model->config.search_alpha);
   }
   model->optimize(simulator, strategies, model->config.search_budget,
       model->config.search_alpha, model->config.computationMode, model->config.enable_propagation);
   if (model->config.export_strategy_file.length() > 0) {
     fprintf(stderr, "Exporting the best discovered strategy to %s.\n",
         model->config.export_strategy_file.c_str());
     std::map<Op*, ParallelConfig>::const_iterator iter;
     std::map<std::string, ParallelConfig> strategy_output;
     for (iter = strategies.begin(); iter != strategies.end(); iter++) {
       strategy_output[iter->first->name] = iter->second;
     }
     save_strategies_to_file(model->config.export_strategy_file, strategy_output);
     fprintf(stderr, "To use the strategy for distributed training, restart"
         " FlexFlow and import the strategy (i.e., --import %s)\n",
         model->config.export_strategy_file.c_str());
     exit(0);
   }  else {
     fprintf(stderr, "The best discovered strategy is not exported.\n"
         "Please set a path to export the strategy using --export or --export-strategy.\n");
     exit(0);
   }
   // Start from data
   // memFBImpl->free_bytes_local(offset, model->config.simulator_work_space_size);
   delete(simulator);
   delete(machine);
 }
 
 
 __host__
 void Simulator::measurement_task(const Task *task,
                                      const std::vector<PhysicalRegion> &regions,
                                      Context ctx, Runtime *runtime)
 {
   const FFModel* model = *((FFModel**) task->args);
   Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
          .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
 
   SimpleMachineModel* nmachine = new SimpleMachineModel(model->config.numNodes, model->config.workersPerNode, gpu_mem.capacity());
   MachineModel *machine;
   machine = reinterpret_cast<MachineModel*>(nmachine);
 
   // Assume this task is running on GPU0
   Simulator* simulator = new Simulator(model, model->handlers[0], gpu_mem, machine);
   // Set cublas/cudnn streams to allow Realm catch the events
 
   hipStream_t stream;
   checkCUDA(get_legion_stream(&stream));
   checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
   checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
 
   const_cast<FFModel*>(model)->measure(simulator);
   
   delete(simulator);
   delete(machine);
 }
 
 
 LogicalTaskgraphBasedSimulator::LogicalTaskgraphBasedSimulator(const FFModel* model,
   FFHandler handler, Memory memory, MachineModel *machine)
 : Simulator(model, handler, memory, machine)
 {
   segment_transfer = true; 
   segment_size = 1024 * 1024; 
 }
 
 __host__
 void LogicalTaskgraphBasedSimulator::simulation_task(const Task *task,
                                      const std::vector<PhysicalRegion> &regions,
                                      Context ctx, Runtime *runtime)
 {
   FFModel* model = *((FFModel**) task->args);
   Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
          .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
   // Realm::MemoryImpl* memImpl =
   //     Realm::get_runtime()->get_memory_impl(gpu_mem);
   // Realm::Cuda::GPUFBMemory* memFBImpl = (Realm::Cuda::GPUFBMemory*) memImpl;
   // off_t offset = memFBImpl->alloc_bytes_local(model->config.simulator_work_space_size);
   // void* base_ptr = memFBImpl->get_direct_ptr(offset, 0);
   // FlatDegConstraintNetworkTopologyGenerator topo_gen = FlatDegConstraintNetworkTopologyGenerator(model->config.numNodes, model->config.node_degree);
   NetworkTopologyGenerator * topo_gen;
   if (model->config.topology == "fattree")
     topo_gen = new BigSwitchNetworkTopologyGenerator(model->config.numNodes);
   else if (model->config.topology == "fc")
     topo_gen = new FCTopologyGenerator(model->config.numNodes);
   else if (model->config.topology == "random") 
     topo_gen = new FlatDegConstraintNetworkTopologyGenerator(model->config.numNodes, model->config.node_degree);
   else 
     assert("Unsupported topology" && false);
   
   NetworkedMachineModel *nmachine = new NetworkedMachineModel(model->config.numNodes, 
     model->config.workersPerNode,  
     model->config.topology == "fattree" ? 1 : 0,
     model->config.network_latency,
     topo_gen->generate_topology(),
     gpu_mem.capacity(),
     model->config.iface_bandwidth
   );
   nmachine->set_pcie(false);
   nmachine->set_pipeline(true);
 
   // SimpleMachineModel* nmachine = new SimpleMachineModel(model->config.numNodes, model->config.workersPerNode, gpu_mem.capacity());
 
   MachineModel *machine;
   machine = reinterpret_cast<MachineModel*>(nmachine);
 
   // Assume this task is running on GPU0
   
   Simulator* simulator = new LogicalTaskgraphBasedSimulator(model, model->handlers[0], gpu_mem, machine);
   if (model->config.mfile != "") {
     model->load_measurement(simulator, model->config.mfile);
   }
   // DemandHeuristicNetworkOptimizer *dhopt = 
   //   new DemandHeuristicNetworkOptimizer(machine);
   // dhopt->if_cnt = model->config.node_degree;
   // simulator->l1optimizer = dhopt;
 
   // Set cublas/cudnn streams to allow Realm catch the events
 
   hipStream_t stream;
   checkCUDA(get_legion_stream(&stream));
   checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
   checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
 
   std::map<Op*, ParallelConfig> strategies;
   if (model->config.import_strategy_file.length() > 0) {
     // Load the strategy from config.strategies
     for (size_t l = 0; l < model->layers.size(); l++) {
       MappingTagID key = FFConfig::get_hash_id(std::string(model->layers[l]->name));
       std::map<MappingTagID, ParallelConfig>::const_iterator iter;
       iter = model->config.strategies.find(key);
       if (iter == model->config.strategies.end()) {
         fprintf(stderr, "ERROR: Cannot find strategy for operator %s in "
                 "strategy file %s\n", model->layers[l]->name,
                 model->config.import_strategy_file.c_str());
       }
       strategies[model->layers[l]] = iter->second;
     }
   } else {
     // Start from data parallel
     int curr_gpu = 0;
     for (size_t l = 0; l < model->layers.size(); l++) {
       // uint64_t opsz = std::numeric_limits<uint64_t>::max();
       // for (int i = 0; i < model->layers[l]->numWeights; i++) {
       //   if (model->layers[l]->weights[i].get_volume() < opsz)  {
       //     opsz = model->layers[l]->weights[i].get_volume() * sizeof(float);
       //   }
       // }
       // if (opsz * model->config.numNodes * model->config.workersPerNode < gpu_mem.capacity())
       if (model->layers[l]->op_type != OperatorType::OP_EMBEDDING)
         strategies[model->layers[l]] = model->layers[l]->get_data_parallel_config(*model);
       else
       {
         ParallelConfig pc = model->layers[l]->get_random_parallel_config(*model);
         pc.device_ids[0] = (curr_gpu++) % model->config.numNodes;
         strategies[model->layers[l]] = pc;
       }
     }
   }
   if (model->config.computationMode == COMP_MODE_TRAINING) {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(TRAINING)\n",
         model->config.search_budget, model->config.search_alpha);
   } else {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(INFERENCE)\n",
         model->config.search_budget, model->config.search_alpha);
   }
   model->optimize(simulator, strategies, model->config.search_budget,
       model->config.search_alpha, model->config.computationMode, model->config.enable_propagation);
   if (model->config.export_strategy_file.length() > 0) {
     fprintf(stderr, "Exporting the best discovered strategy to %s.\n",
         model->config.export_strategy_file.c_str());
     std::map<Op*, ParallelConfig>::const_iterator iter;
     std::map<std::string, ParallelConfig> strategy_output;
     for (iter = strategies.begin(); iter != strategies.end(); iter++) {
       strategy_output[iter->first->name] = iter->second;
     }
     save_strategies_to_file(model->config.export_strategy_file, strategy_output);
     fprintf(stderr, "To use the strategy for distributed training, restart"
         " FlexFlow and import the strategy (i.e., --import %s)\n",
         model->config.export_strategy_file.c_str());
     exit(0);
   }  else {
     fprintf(stderr, "The best discovered strategy is not exported.\n"
         "Please set a path to export the strategy using --export or --export-strategy.\n");
     exit(0);
   }
   // Start from data
   // memFBImpl->free_bytes_local(offset, model->config.simulator_work_space_size);
   delete(simulator);
   delete(machine);
 }
 
 SpMulMatSimulator::SpMulMatSimulator(const FFModel* model,
   FFHandler handler, Memory memory, MachineModel *machine) 
 : LogicalTaskgraphBasedSimulator(model, handler, memory, machine)
 {
 
 }
 
 
 __host__
 void SpMulMatSimulator::simulation_task(const Task *task,
                                      const std::vector<PhysicalRegion> &regions,
                                      Context ctx, Runtime *runtime)
 {
   FFModel* model = *((FFModel**) task->args);
   Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
          .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
   // Realm::MemoryImpl* memImpl =
   //     Realm::get_runtime()->get_memory_impl(gpu_mem);
   // Realm::Cuda::GPUFBMemory* memFBImpl = (Realm::Cuda::GPUFBMemory*) memImpl;
   // off_t offset = memFBImpl->alloc_bytes_local(model->config.simulator_work_space_size);
   // void* base_ptr = memFBImpl->get_direct_ptr(offset, 0);
   // FlatDegConstraintNetworkTopologyGenerator topo_gen = FlatDegConstraintNetworkTopologyGenerator(model->config.numNodes, model->config.node_degree);
   // BigSwitchNetworkTopologyGenerator topo_gen = BigSwitchNetworkTopologyGenerator(model->config.numNodes);
   FlatEmptyNetworkTopologyGenerator topo_gen = FlatEmptyNetworkTopologyGenerator(model->config.numNodes);
   
   NetworkedMachineModel *nmachine = new NetworkedMachineModel(model->config.numNodes, 
     model->config.workersPerNode,  
     0,
     model->config.network_latency,
     topo_gen.generate_topology(),
     gpu_mem.capacity(),
     model->config.iface_bandwidth
   );
   nmachine->set_pcie(false);
   nmachine->set_pipeline(true);
 
   // SimpleMachineModel* nmachine = new SimpleMachineModel(model->config.numNodes, model->config.workersPerNode, gpu_mem.capacity());
 
   MachineModel *machine;
   machine = reinterpret_cast<MachineModel*>(nmachine);
 
   // Assume this task is running on GPU0
   
   Simulator* simulator = new SpMulMatSimulator(model, model->handlers[0], gpu_mem, machine);
   if (model->config.mfile != "") {
     model->load_measurement(simulator, model->config.mfile);
   }
   SpMulMat *dhopt = 
     new SpMulMat(machine, model->config.node_degree, true);
   // dhopt->if_cnt = model->config.node_degree;
   simulator->l1optimizer = dhopt;
 
   // Set cublas/cudnn streams to allow Realm catch the events
 
   if (!model->config.nogpu) {
     hipStream_t stream;
     checkCUDA(get_legion_stream(&stream));
     checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
     checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
   }
 
   std::map<Op*, ParallelConfig> strategies;
   if (model->config.import_strategy_file.length() > 0) {
     // Load the strategy from config.strategies
     for (size_t l = 0; l < model->layers.size(); l++) {
       MappingTagID key = FFConfig::get_hash_id(std::string(model->layers[l]->name));
       std::map<MappingTagID, ParallelConfig>::const_iterator iter;
       iter = model->config.strategies.find(key);
       if (iter == model->config.strategies.end()) {
         fprintf(stderr, "ERROR: Cannot find strategy for operator %s in "
                 "strategy file %s\n", model->layers[l]->name,
                 model->config.import_strategy_file.c_str());
       }
       strategies[model->layers[l]] = iter->second;
     }
   } else {
     // Start from data parallel
     int curr_gpu = 0;
     for (size_t l = 0; l < model->layers.size(); l++) {
       // uint64_t opsz = std::numeric_limits<uint64_t>::max();
       // for (int i = 0; i < model->layers[l]->numWeights; i++) {
       //   if (model->layers[l]->weights[i].get_volume() < opsz)  {
       //     opsz = model->layers[l]->weights[i].get_volume() * sizeof(float);
       //   }
       // }
       // if (opsz * model->config.numNodes * model->config.workersPerNode < gpu_mem.capacity())
       if (model->layers[l]->op_type != OperatorType::OP_EMBEDDING)
         strategies[model->layers[l]] = model->layers[l]->get_data_parallel_config(*model);
         //strategies[model->layers[l]] = model->layers[l]->get_random_parallel_config(*model);
       else {
         ParallelConfig pc = model->layers[l]->get_random_parallel_config(*model);
         pc.device_ids[0] = (curr_gpu++) % model->config.numNodes;
         strategies[model->layers[l]] = pc;
       }
     }
 
   }
   if (model->config.computationMode == COMP_MODE_TRAINING) {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(TRAINING)\n",
         model->config.search_budget, model->config.search_alpha);
   } else {
     fprintf(stderr, "MCMC search configuration: budget(%zu) alpha(%.8lf) mode(INFERENCE)\n",
         model->config.search_budget, model->config.search_alpha);
   }
   model->optimize(simulator, strategies, model->config.search_budget,
       model->config.search_alpha, model->config.computationMode, model->config.enable_propagation);
   if (model->config.export_strategy_file.length() > 0) {
     fprintf(stderr, "Exporting the best discovered strategy to %s.\n",
         model->config.export_strategy_file.c_str());
     std::map<Op*, ParallelConfig>::const_iterator iter;
     std::map<std::string, ParallelConfig> strategy_output;
     for (iter = strategies.begin(); iter != strategies.end(); iter++) {
       strategy_output[iter->first->name] = iter->second;
     }
     save_strategies_to_file(model->config.export_strategy_file, strategy_output);
     fprintf(stderr, "To use the strategy for distributed training, restart"
         " FlexFlow and import the strategy (i.e., --import %s)\n",
         model->config.export_strategy_file.c_str());
     exit(0);
   }  else {
     fprintf(stderr, "The best discovered strategy is not exported.\n"
         "Please set a path to export the strategy using --export or --export-strategy.\n");
     exit(0);
   }
   // Start from data
   // memFBImpl->free_bytes_local(offset, model->config.simulator_work_space_size);
   delete(simulator);
   delete(machine);
 }
 
